#include "hip/hip_runtime.h"
#include "unary.cuh"

template <class T>
static __global__ void neg(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = -x[i];
}

template <class T>
static __global__ void step(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = x[i] > (T)0.0f;
}

template <class T>
static __global__ void gelu(const T * x, T * dst, const int k) {
    const T GELU_COEF_A    = 0.044715f;
    const T SQRT_2_OVER_PI = 0.79788456080286535587989211986876f;
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    T xi = x[i];
    dst[i] = (T)0.5f*xi*((T)1.0f + (T)tanhf(SQRT_2_OVER_PI*xi*((T)1.0f + GELU_COEF_A*xi*xi)));
}

template <class T>
static __global__ void gelu_quick(const T * x, T * dst, int k) {
    const T GELU_QUICK_COEF = -1.702f;
    const int i  = blockDim.x*blockIdx.x + threadIdx.x;
    if (i >= k) {
        return;
    }
    dst[i] = x[i] * ((T)1.0f / ((T)1.0f + (T)expf(GELU_QUICK_COEF * x[i])));
}

template <class T>
static __global__ void silu(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] / ((T)1.0f + (T)expf(-x[i]));
}

template <class T>
static __global__ void silu_back(
        const T * grad, const T * xf, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    const T xfi = xf[i];
    const T s = (T)1.0f / ((T)1.0f + (T)expf(-xfi));
    dst[i] = grad[i] * s * ((T)1.0f + xfi * ((T)1.0f - s));
}

template <class T>
static __global__ void tanh(const T * x, T * dst, int k) {
    const int i  = blockDim.x*blockIdx.x + threadIdx.x;
    if (i >= k) {
        return;
    }
    dst[i] = tanhf(x[i]);
}

template <class T>
static __global__ void relu(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = fmaxf(x[i], 0);
}

template <class T>
static __global__ void sigmoid(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = (T)1.0f / ((T)1.0f + (T)expf(-x[i]));
}

template <class T>
static __global__ void hardsigmoid(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = fminf(1.0f, fmaxf(0.0f, (x[i] + (T)3.0f) / (T)6.0f));
}

template <class T>
static __global__ void hardswish(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] * (T)fminf(1.0f, fmaxf(0.0f, (x[i] + (T)3.0f) / (T)6.0f));
}

template <class T>
static __global__ void exp(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = expf(x[i]);
}

template <class T>
static __global__ void leaky_relu(const T * x, T * dst, const int k, const float negative_slope) {
    const int i  = blockDim.x*blockIdx.x + threadIdx.x;
    if (i >= k) {
        return;
    }
    dst[i] = (T)fmaxf(x[i], 0) + (T)fminf(x[i], 0.0f) * (T)negative_slope;
}

template <class T>
static __global__ void sqr(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] * x[i];
}

template <class T>
static __global__ void sqrt(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = sqrtf(x[i]);
}

template <class T>
static __global__ void sin(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = sinf(x[i]);
}

template <class T>
static __global__ void cos(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = cosf(x[i]);
}

template <class T>
static void neg_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_NEG_BLOCK_SIZE - 1) / CUDA_NEG_BLOCK_SIZE;
    neg<<<num_blocks, CUDA_NEG_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void step_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_STEP_BLOCK_SIZE - 1) / CUDA_STEP_BLOCK_SIZE;
    step<<<num_blocks, CUDA_STEP_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void gelu_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_GELU_BLOCK_SIZE - 1) / CUDA_GELU_BLOCK_SIZE;
    gelu<<<num_blocks, CUDA_GELU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void gelu_quick_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_GELU_BLOCK_SIZE - 1) / CUDA_GELU_BLOCK_SIZE;
    gelu_quick<<<num_blocks, CUDA_GELU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void silu_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SILU_BLOCK_SIZE - 1) / CUDA_SILU_BLOCK_SIZE;
    silu<<<num_blocks, CUDA_SILU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void silu_back_cuda(const T * grad, const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SILU_BACK_BLOCK_SIZE - 1) / CUDA_SILU_BLOCK_SIZE;
    silu_back<<<num_blocks, CUDA_SILU_BACK_BLOCK_SIZE, 0, stream>>>(grad, x, dst, k);
}

template <class T>
static void tanh_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_TANH_BLOCK_SIZE - 1) / CUDA_TANH_BLOCK_SIZE;
    tanh<<<num_blocks, CUDA_TANH_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void relu_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_RELU_BLOCK_SIZE - 1) / CUDA_RELU_BLOCK_SIZE;
    relu<<<num_blocks, CUDA_RELU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void sigmoid_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SIGMOID_BLOCK_SIZE - 1) / CUDA_SIGMOID_BLOCK_SIZE;
    sigmoid<<<num_blocks, CUDA_SIGMOID_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void hardsigmoid_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_HARDSIGMOID_BLOCK_SIZE - 1) / CUDA_HARDSIGMOID_BLOCK_SIZE;
    hardsigmoid<<<num_blocks, CUDA_HARDSIGMOID_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void hardswish_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_HARDSWISH_BLOCK_SIZE - 1) / CUDA_HARDSWISH_BLOCK_SIZE;
    hardswish<<<num_blocks, CUDA_HARDSWISH_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void exp_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_EXP_BLOCK_SIZE - 1) / CUDA_EXP_BLOCK_SIZE;
    exp<<<num_blocks, CUDA_EXP_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void leaky_relu_cuda(const T * x, T * dst, const int k, const float negative_slope, hipStream_t stream) {
    const int num_blocks = (k + CUDA_RELU_BLOCK_SIZE - 1) / CUDA_RELU_BLOCK_SIZE;
    leaky_relu<<<num_blocks, CUDA_RELU_BLOCK_SIZE, 0, stream>>>(x, dst, k, negative_slope);
}

template <class T>
static void sqr_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SQR_BLOCK_SIZE - 1) / CUDA_SQR_BLOCK_SIZE;
    sqr<<<num_blocks, CUDA_SQR_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void sqrt_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SQRT_BLOCK_SIZE - 1) / CUDA_SQRT_BLOCK_SIZE;
    sqrt<<<num_blocks, CUDA_SQRT_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void sin_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SIN_BLOCK_SIZE - 1) / CUDA_SIN_BLOCK_SIZE;
    sin<<<num_blocks, CUDA_SIN_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <class T>
static void cos_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_COS_BLOCK_SIZE - 1) / CUDA_COS_BLOCK_SIZE;
    cos<<<num_blocks, CUDA_COS_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

void ggml_cuda_op_neg(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        neg_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        neg_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_step(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        step_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        step_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_gelu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        gelu_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        gelu_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_silu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        silu_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        silu_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_silu_back(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0]; // input from forward pass
    const ggml_tensor * src1 = dst->src[1]; // grads of forward pass output

    const float * src0_d = (const float *) src0->data;
    const float * src1_d = (const float *) src1->data;
    float       * dst_d  = (float       *) dst->data;

    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        silu_back_cuda((const half *)src0_d, (const half *)src1_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        silu_back_cuda((const float*)src0_d, (const float*)src1_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_gelu_quick(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        gelu_quick_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        gelu_quick_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_tanh(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        tanh_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        tanh_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_relu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        relu_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        relu_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_sigmoid(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        sigmoid_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        sigmoid_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_hardsigmoid(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        hardsigmoid_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        hardsigmoid_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_hardswish(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        hardswish_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        hardswish_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_exp(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        exp_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        exp_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_leaky_relu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    float negative_slope;
    memcpy(&negative_slope, dst->op_params, sizeof(float));

    if (src0->type == GGML_TYPE_F16) {
        leaky_relu_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), negative_slope, stream);
    } else {
        leaky_relu_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), negative_slope, stream);
    }
}

void ggml_cuda_op_sqr(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        sqr_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        sqr_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_sqrt(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        sqrt_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        sqrt_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_sin(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        sin_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        sin_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_cos(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        cos_cuda((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        cos_cuda((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}
